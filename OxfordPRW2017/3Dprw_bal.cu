/*
 3D PRW_bal CUDA
 Copyright Spatial Reasoning Group,
 Department of Computer Science, University of Oxford, 2017.
 All rights reserved.
*/


#include <hip/hip_runtime.h>
#include <stdio.h>


////////////////////////////////////////////////////////////////////////
// define kernel block size
////////////////////////////////////////////////////////////////////////

#define BLOCK_X 8
#define BLOCK_Y 8
#define BLOCK_Z 8
#define BLOCK_EX (BLOCK_X+2)
#define BLOCK_EY (BLOCK_Y+2)
#define BLOCK_EZ (BLOCK_Z+2)
#define BLOCK_EXY ((BLOCK_EX)*(BLOCK_EY))
#define BLOCK ((BLOCK_EX)*(BLOCK_EY)*(BLOCK_EZ))

#define RR 6


__device__ bool change = false;


__global__ void initialise_state(int NX, int NY, int NZ,
						const unsigned char *d_array,
										int *d_label,
										int *d_state)
{
	// pixel has lower neighbour - 0, is local minimum - 1,
	// some neighbours equal and some higher - 2 or 3

	int i, j, k, ind, NXY,
		min, state,
		nind[6];
	unsigned char minarray;
	bool ncond[6];

	// define global indices
	i = threadIdx.x + blockIdx.x*blockDim.x;
	j = threadIdx.y + blockIdx.y*blockDim.y;
	k = threadIdx.z + blockIdx.z*blockDim.z;

	if (i < NX && j < NY && k < NZ) // && i >= 0 && j >= 0 && k >= 0
	{
		NXY = NX*NY;

		ind = i + j*NX + k*NXY;

		// initialise neighbour indices and conditions
		nind[0] = ind - NXY;	ncond[0] = k > 0;
		nind[1] = ind - NX;		ncond[1] = j > 0;
		nind[2] = ind - 1;		ncond[2] = i > 0;
		nind[3] = ind + 1;		ncond[3] = i + 1 < NX;
		nind[4] = ind + NX;		ncond[4] = j + 1 < NY;
		nind[5] = ind + NXY;	ncond[5] = k + 1 < NZ;

		// define initial state
		minarray = d_array[ind];
		min = ind;
		state = 1; // assume local minimum

		for (int n = 0; n < 6; ++n)
			if (ncond[n] && d_array[nind[n]] <= minarray)
			{
				minarray = d_array[nind[n]];
				min = nind[n];
			}
		if (minarray < d_array[ind]) // lower neighbour
			state = 0;
		else if (min > ind) // equal neighbour with larger index, no lower neighbour
			state = 2;
		else if (min < ind) // equal neighbour with smaller index, no lower neighbour
		{
			state = 3;
			min = ind;
		}

		d_label[ind] = min;
		d_state[ind] = state;
	}
}



__global__ void resolve_plateaux(int NX, int NY, int NZ,
						const unsigned char *d_array,
										int *d_label,
								  const int *d_state,
										int *d_newstate)
{
	// resolve non-minimal plateaux changing states from 2, 3 to 0

	int i, j, k, si, sj, sk, ind, sind, NXY,
		label, state, nstate,
		nind[6], snind[6];
	bool valid,
		ncond[6], sbcond[6];

	__shared__ unsigned char s_array[BLOCK];
	__shared__ int s_state[BLOCK];
	__shared__ bool s_change;

	// define global indices
	i = threadIdx.x + blockIdx.x*blockDim.x;
	j = threadIdx.y + blockIdx.y*blockDim.y;
	k = threadIdx.z + blockIdx.z*blockDim.z;
	si = threadIdx.x + 1;
	sj = threadIdx.y + 1;
	sk = threadIdx.z + 1;

	valid = i < NX && j < NY && k < NZ; // && i >= 0 && j >= 0 && k >= 0

	if (valid)
	{
		NXY = NX*NY;

		ind = i + j*NX + k*NXY;
		sind = si + sj*BLOCK_EX + sk*BLOCK_EXY;

		// initialise neighbour indices and conditions
		nind[0] = ind - NXY;			ncond[0] = k > 0;
		nind[1] = ind - NX;				ncond[1] = j > 0;
		nind[2] = ind - 1;				ncond[2] = i > 0;
		nind[3] = ind + 1;				ncond[3] = i + 1 < NX;
		nind[4] = ind + NX;				ncond[4] = j + 1 < NY;
		nind[5] = ind + NXY;			ncond[5] = k + 1 < NZ;
		snind[0] = sind - BLOCK_EXY;	sbcond[0] = sk == 1;
		snind[1] = sind - BLOCK_EX;		sbcond[1] = sj == 1;
		snind[2] = sind - 1;			sbcond[2] = si == 1;
		snind[3] = sind + 1;			sbcond[3] = si == BLOCK_X;
		snind[4] = sind + BLOCK_EX;		sbcond[4] = sj == BLOCK_Y;
		snind[5] = sind + BLOCK_EXY;	sbcond[5] = sk == BLOCK_Z;

		label = d_label[ind];

		// copy data into shared memory, including extra border
		s_array[sind] = d_array[ind];
		s_state[sind] = d_state[ind];
		for (int n = 0; n < 6; ++n)
			if (sbcond[n] && ncond[n])
			{
				s_array[snind[n]] = d_array[nind[n]];
				s_state[snind[n]] = d_state[nind[n]];
			}
	}

	do
	{
		__syncthreads();
		s_change = false;
		__syncthreads();

		if (valid)
		{
			state = s_state[sind];

			if (state >= 2 || state < 0)
			{
				for (int n = 0; n < 6; ++n)
				{
					nstate = s_state[snind[n]] - 1;
					if (ncond[n] && nstate < 0 && s_array[snind[n]] == s_array[sind] && (state >= 2 || state < nstate))
					{
						state = nstate;
						label = nind[n];
					}
				}
				// if state changed update shared and global change variables
				if (state != s_state[sind])
				{
					s_change = true;
					change = true;
				}
			}
		}

		__syncthreads();
		if (valid)
			s_state[sind] = state;
	} while (s_change);

	if (valid)
	{
		d_label[ind] = label;
		d_newstate[ind] = s_state[sind];
	}
}



__global__ void propagate_labels(int NX, int NY, int NZ,
								int *d_label)
{
	// propagate labels uphill updating current from its label

	int i, j, k, ind,
		newlab, oldlab;

	// define global indices
	i = threadIdx.x + blockIdx.x*blockDim.x;
	j = threadIdx.y + blockIdx.y*blockDim.y;
	k = threadIdx.z + blockIdx.z*blockDim.z;

	if (i < NX && j < NY && k < NZ) // && i >= 0 && j >= 0 && k >= 0
	{
		ind = i + j*NX + k*NX*NY;

		oldlab = ind;
		newlab = d_label[ind];
		for (int n = 0; newlab != oldlab && n < RR; ++n)
		{
			oldlab = newlab;
			newlab = d_label[oldlab];
		}
		d_label[ind] = newlab;
		if (newlab != oldlab)
			change = true;
	}
}



__global__ void unify_plateau_labels(int NX, int NY, int NZ,
									  int *d_label,
								const int *d_state)
{
	// unify multiple labels within minimal plateaux

	int i, j, k, ind, NXY,
		indlab, neilab,
		nind[6];
	bool ncond[6];

	// define global indices
	i = threadIdx.x + blockIdx.x*blockDim.x;
	j = threadIdx.y + blockIdx.y*blockDim.y;
	k = threadIdx.z + blockIdx.z*blockDim.z;

	if (i < NX && j < NY && k < NZ) // && i >= 0 && j >= 0 && k >= 0
	{
		NXY = NX*NY;

		ind = i + j*NX + k*NXY;

		if (d_state[ind] >= 2)
		{
			// initialise neighbour indices and conditions
			nind[0] = ind - NXY;	ncond[0] = k > 0;
			nind[1] = ind - NX;		ncond[1] = j > 0;
			nind[2] = ind - 1;		ncond[2] = i > 0;
			nind[3] = ind + 1;		ncond[3] = i + 1 < NX;
			nind[4] = ind + NX;		ncond[4] = j + 1 < NY;
			nind[5] = ind + NXY;	ncond[5] = k + 1 < NZ;

			for (int n = 0; n < 6; ++n)
				if (ncond[n] && d_state[nind[n]] >= 2)
				{
					indlab = d_label[ind];
					neilab = d_label[nind[n]];
					while (d_label[indlab] != d_label[neilab])
					{
						atomicMin(&d_label[indlab], d_label[neilab]);
						atomicMin(&d_label[neilab], d_label[indlab]);
						change = true;
					}
				}
		}
		while (d_label[ind] != d_label[d_label[ind]])
			d_label[ind] = d_label[d_label[ind]];
	}
}




// 'h_' prefix - CPU (host) memory space
unsigned char *h_array;
int *h_label;

int	NX = 1, NY = 1, NZ = 1, NS;


void mywatershed()
{
	// 'h_' prefix - CPU (host) memory space
	int bx, by, bz;
//	int counter;
	bool h_change;

	// 'd_' prefix - GPU (device) memory space
	unsigned char *d_array;
	int *d_label;
	int *d_state, *d_new, *d_temp;


	// initialise CUDA timing
	float milli;
	hipEvent_t overallstart, stop;
	hipEventCreate(&overallstart);
//	cudaEventCreate(&start);
	hipEventCreate(&stop);

	// start the overall timer
	hipEventRecord(overallstart);

	// allocate memory for arrays
	hipMalloc((void **)&d_array,	NS);
	hipMalloc((void **)&d_label,	sizeof(int)*NS);
	hipMalloc((void **)&d_state,	sizeof(int)*NS);
	hipMalloc((void **)&d_new,		sizeof(int)*NS);


	// copy array to device
//	cudaEventRecord(start);
	hipMemcpy(d_array, h_array, NS, hipMemcpyHostToDevice);
//	cudaEventRecord(stop);
//	cudaEventSynchronize(stop);
//	cudaEventElapsedTime(&milli, start, stop);
//	printf("\nCopy array to device: %.3f (ms) \n", milli);

	// Set up the execution configuration
	bx = 1 + (NX-1)/BLOCK_X;
	by = 1 + (NY-1)/BLOCK_Y;
	bz = 1 + (NZ-1)/BLOCK_Z;

	dim3 dimGrid(bx,by,bz);
	dim3 dimBlock(BLOCK_X,BLOCK_Y,BLOCK_Z);

//	printf("\n dimGrid = %d %d %d \n",dimGrid.x,dimGrid.y,dimGrid.z);
//	printf(" dimBlock = %d %d %d \n",dimBlock.x,dimBlock.y,dimBlock.z);


	// Execute GPU kernel to initialise state
//	cudaEventRecord(start);

	initialise_state<<<dimGrid, dimBlock>>>(NX, NY, NZ, d_array, d_label, d_state);

//	cudaEventRecord(stop);
//	cudaEventSynchronize(stop);
//	cudaEventElapsedTime(&milli, start, stop);
//	printf("initialise_state: %.3f (ms) \n", milli);


	// Execute GPU kernel to resolve plateaux
//	cudaEventRecord(start);

//	counter = 0;
	h_change = true;
	while (h_change)
	{
//		++counter;
		h_change = false;
		hipMemcpyToSymbol(HIP_SYMBOL(change), &h_change, sizeof(h_change));

		resolve_plateaux<<<dimGrid, dimBlock>>>(NX, NY, NZ, d_array, d_label, d_state, d_new);

		d_temp = d_state; d_state = d_new; d_new = d_temp;
		hipMemcpyFromSymbol(&h_change, HIP_SYMBOL(change), sizeof(change));
	}

//	cudaEventRecord(stop);
//	cudaEventSynchronize(stop);
//	cudaEventElapsedTime(&milli, start, stop);
//	printf("resolve_plateaux: %.3f (ms) \n", milli);
//	printf("resolve_plateaux number of iterations = %d\n", counter);


	// Execute GPU kernel to propagate labels (run the CA iterations)
//	cudaEventRecord(start);

//	counter = 0;
	h_change = true;
	while (h_change)
	{
//		++counter;
		h_change = false;
		hipMemcpyToSymbol(HIP_SYMBOL(change), &h_change, sizeof(h_change));

		propagate_labels<<<dimGrid, dimBlock>>>(NX, NY, NZ, d_label);

		hipMemcpyFromSymbol(&h_change, HIP_SYMBOL(change), sizeof(change));
	}

//	cudaEventRecord(stop);
//	cudaEventSynchronize(stop);
//	cudaEventElapsedTime(&milli, start, stop);
//	printf("propagate_labels: %.3f (ms) \n", milli);
//	printf("propagate_labels number of iterations = %d\n", counter);


	// Execute GPU kernel to unify minimal plateau labels
//	cudaEventRecord(start);

//	counter = 0;
	h_change = true;
	while (h_change)
	{
//		++counter;
		h_change = false;
		hipMemcpyToSymbol(HIP_SYMBOL(change), &h_change, sizeof(h_change));

		unify_plateau_labels<<<dimGrid, dimBlock>>>(NX, NY, NZ, d_label, d_state);

		hipMemcpyFromSymbol(&h_change, HIP_SYMBOL(change), sizeof(change));
	}

//	cudaEventRecord(stop);
//	cudaEventSynchronize(stop);
//	cudaEventElapsedTime(&milli, start, stop);
//	printf("unify_plateau_labels: %.3f (ms) \n", milli);
//	printf("unify_plateau_labels number of iterations = %d\n", counter);


	// Read back GPU results
//	cudaEventRecord(start);
	hipMemcpy(h_label, d_label, sizeof(int)*NS, hipMemcpyDeviceToHost);
//	cudaEventRecord(stop);
//	cudaEventSynchronize(stop);
//	cudaEventElapsedTime(&milli, start, stop);
//	printf("\nCopy label to host: %.3f (ms) \n", milli);


	// Release GPU memory
	hipFree(d_array);
	hipFree(d_label);
	hipFree(d_state);
	hipFree(d_new);

	// stop the overall timer
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milli, overallstart, stop);
	printf("\n\nOverall execution time: %.3f (ms) \n", milli);
}



int main(int argc, const char **argv)
{
	NX = 10;
	NY = 10;
	NS = NX*NY*NZ;

	// allocate memory for label array
	h_label = (int *)malloc(sizeof(int)*NS);
	h_array = (unsigned char *)malloc(NS);

	for (int i = 0; i < NS; ++i)
		if (i == 11 || i == 18 || i == 81 || i == 88)
			h_array[i] = 0;
		else
			h_array[i] = 1;

	mywatershed();

	for (int j = 0; j < NY; ++j)
	{
		for (int i = 0; i < NX; ++i)
			printf("%d ", h_label[i + j*NX]);
		printf("\n");
	}

	// Release CPU memory
	free(h_array);
	free(h_label);

	return 0;
}
